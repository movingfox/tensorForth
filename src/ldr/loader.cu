
#include <hip/hip_runtime.h>
/** -*- c++ -*-
 * @file
 * @brief Loader class - dataset loader factory implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#if (T4_ENABLE_OBJ && T4_ENABLE_NN)
#include <map>
#include "mnist.h"
#include "loader.h"
///
/// Note:
///   const char* key in map will not work because ptr1 != ptr2
///   but the string conversion slows it down by 3x.
///   We have only a few, so not too bad. Also we cache top <=> dataset
///
typedef std::map<std::string, Corpus*> CorpusMap;
typedef std::map<int, Corpus*> DsetMap;
CorpusMap cp_map;                          ///< string name, Corpus pair
DsetMap   ds_map;                          ///< Dataset, Corpus pair (cache)
///
/// TODO: to read from YAML config file
///
void Loader::init(bool trace) {
    cp_map["mnist_train"] =
        new Mnist(
            "../data/MNIST/raw/train-images-idx3-ubyte",
            "../data/MNIST/raw/train-labels-idx1-ubyte", trace);
    cp_map["mnist_test"] =
        new Mnist(
            "../data/MNIST/raw/t10k-images-idx3-ubyte",
            "../data/MNIST/raw/t10k-labels-idx1-ubyte", trace);
}

Corpus *Loader::get(int dset, const char *ds_name) {
    DsetMap::iterator dsi = ds_map.find(dset);          /// * cache hit?
    if (dsi != ds_map.end()) return dsi->second;

    if (!ds_name) return NULL;                          /// * no name given

    CorpusMap::iterator cpi = cp_map.find(ds_name);     /// * create new entry
    if (cpi == cp_map.end()) return NULL;

    return ds_map[dset] = cpi->second;
}

#endif // (T4_ENABLE_OBJ && T4_ENABLE_NN)


